#include "hip/hip_runtime.h"
/*
 * @file  cuCorrFrequency.cu
 * @brief A class performs cross correlation in frequency domain
 */

#include "cuCorrFrequency.h"
#include "cuAmpcorUtil.h"

/*
 * cuFreqCorrelator Constructor
 * @param imageNX height of each image
 * @param imageNY width of each image
 * @param nImages number of images in the batch
 * @param stream CUDA stream
 */
cuFreqCorrelator::cuFreqCorrelator(int imageNX, int imageNY, int nImages, hipStream_t stream_)
{

    int imageSize = imageNX*imageNY;
    int fImageSize = imageNX*(imageNY/2+1);
    int n[NRANK] ={imageNX, imageNY};
    
    // set up fft plans
    cufft_Error(hipfftPlanMany(&forwardPlan, NRANK, n,
                              NULL, 1, imageSize,
                              NULL, 1, fImageSize, 
                              HIPFFT_R2C, nImages));
    cufft_Error(hipfftPlanMany(&backwardPlan, NRANK, n, 
                              NULL, 1, fImageSize,
                              NULL, 1, imageSize, 
                              HIPFFT_C2R, nImages));
    stream = stream_;
    hipfftSetStream(forwardPlan, stream);
    hipfftSetStream(backwardPlan, stream);

    // set up work arrays
    workFM = new cuArrays<float2>(imageNX, (imageNY/2+1), nImages);
    workFM->allocate();
    workFS = new cuArrays<float2>(imageNX, (imageNY/2+1), nImages);
    workFS->allocate();
    workT = new cuArrays<float> (imageNX, imageNY, nImages);
    workT->allocate();
}

/// destructor
cuFreqCorrelator::~cuFreqCorrelator()
{
    cufft_Error(hipfftDestroy(forwardPlan));
    cufft_Error(hipfftDestroy(backwardPlan));	
    workFM->deallocate();
    workFS->deallocate();
    workT->deallocate();
}	


/**
 * Execute the cross correlation
 * @param[in] templates the reference windows
 * @param[in] images the search windows
 * @param[out] results the correlation surfaces
 */

void cuFreqCorrelator::execute(cuArrays<float> *templates, cuArrays<float> *images, cuArrays<float> *results)
{
    // pad the reference windows to the the size of search windows
    cuArraysCopyPadded(templates, workT, stream);
    // forward fft to frequency domain
    cufft_Error(hipfftExecR2C(forwardPlan, workT->devData, workFM->devData));
    cufft_Error(hipfftExecR2C(forwardPlan, images->devData, workFS->devData));
    // cufft doesn't normalize, so manually get the image size for normalization
    float coef = 1.0/(images->size);
    // multiply reference with secondary windows in frequency domain
    cuArraysElementMultiplyConjugate(workFM, workFS, coef, stream);
    // backward fft to get correlation surface in time domain
    cufft_Error(hipfftExecC2R(backwardPlan, workFM->devData, workT->devData));
    // extract to get proper size of correlation surface
    cuArraysCopyExtract(workT, results, make_int2(0, 0), stream);
    // all done
}

// a = a^* * b
inline __device__ float2 cuMulConj(float2 a, float2 b)
{
    return make_float2(a.x*b.x + a.y*b.y, -a.y*b.x + a.x*b.y);
}

// cuda kernel for cuArraysElementMultiplyConjugate
__global__ void cudaKernel_elementMulConjugate(float2 *ainout, float2 *bin, int size, float coef)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < size) {
        hipComplex prod; 
        prod = cuMulConj(ainout[idx], bin[idx]);
        ainout [idx] = prod*coef;
    }
} 

/**
 * Perform multiplication of coef*Conjugate[image1]*image2 for each element
 * @param[inout] image1, the first image
 * @param[in] image2, the secondary image
 * @param[in] coef, usually the normalization factor
 */
void cuArraysElementMultiplyConjugate(cuArrays<float2> *image1, cuArrays<float2> *image2, float coef, hipStream_t stream)
{
    int size = image1->getSize();
    int threadsperblock = NTHREADS;
    int blockspergrid = IDIVUP (size, threadsperblock);
    cudaKernel_elementMulConjugate<<<blockspergrid, threadsperblock, 0, stream>>>(image1->devData, image2->devData, size, coef );
    getLastCudaError("cuArraysElementMultiply error\n");
} 
//end of file
