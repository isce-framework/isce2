#include "hip/hip_runtime.h"
/*
 * @file  cuDeramp.cu
 * @brief Derampling a batch of 2D complex images with GPU
 *
 * A phase ramp is equivalent to a frequency shift in frequency domain,
 *   which needs to be removed (deramping) in order to move the band center
 *   to zero. This is necessary before oversampling a complex signal.
 * Method 1: each signal is decomposed into real and imaginary parts,
 *   and the average phase shift is obtained as atan(\sum imag / \sum real).
 *   The average is weighted by the amplitudes (coherence).
 * Method 0 or else: skip deramping
 *
 */
 
#include "cuArrays.h" 
#include "float2.h"
#include "data_types.h"
#include "hipError_t.h"
#include "cudaUtil.h"
#include "cuAmpcorUtil.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <limits>


// cuda does not have a good support on volatile vector struct, e.g. real2_type
// have to use regular real_type type for shared memory (volatile) data
// the following methods are defined to operate real2_type/complex objects through real_type
inline static __device__ void copyToShared(volatile real_type *s, const int i, const real2_type x, const int block) 
{ s[i] = x.x; s[i+block] = x.y; }

inline static __device__ void copyFromShared(real2_type &x, volatile real_type *s, const int i, const int block) 
{ x.x = s[i]; x.y = s[i+block]; }


inline static __device__ void addInShared(volatile real_type *s, const int i, const int j, const int block) 
{ s[i] += s[i+j]; s[i+block] += s[i+j+block];}


// kernel to do sum reduction for real2_type within a block
template <const int nthreads>
__device__ void complexSumReduceBlock(real2_type& sum, volatile real_type *shmem)
{
    const int tid = threadIdx.x;
    copyToShared(shmem, tid, sum, nthreads);
    __syncthreads();
    
    if (nthreads >=1024) { if (tid < 512) { addInShared(shmem, tid, 512, nthreads); } __syncthreads(); }
    if (nthreads >= 512) { if (tid < 256) { addInShared(shmem, tid, 256, nthreads); } __syncthreads(); }
    if (nthreads >= 256) { if (tid < 128) { addInShared(shmem, tid, 128, nthreads); } __syncthreads(); }
    if (nthreads >= 128) { if (tid <  64) { addInShared(shmem, tid,  64, nthreads); } __syncthreads(); }
    if (tid < 32)
    {	
        addInShared(shmem, tid, 32, nthreads);
        addInShared(shmem, tid, 16, nthreads);
        addInShared(shmem, tid,  8, nthreads);
        addInShared(shmem, tid,  4, nthreads);
        addInShared(shmem, tid,  2, nthreads);
        addInShared(shmem, tid,  1, nthreads); 
    }
    __syncthreads();
    copyFromShared(sum, shmem, 0, nthreads);
}

// cuda kernel for cuDerampMethod1
template<const int nthreads>
__global__ void cuDerampMethod1_kernel(real2_type *images, const int imageNX, int const imageNY, 
    const int imageSize, const int nImages, const real_type normCoef)
{
    __shared__ real_type shmem[2*nthreads];
    int pixelIdx, pixelIdxX, pixelIdxY;
    
    const int bid = blockIdx.x;    
    if(bid >= nImages) return;
    real2_type *image = images+ bid*imageSize;
    const int tid = threadIdx.x;  
    real2_type phaseDiffY  = make_real2(0.0, 0.0);
    for (int i = tid; i < imageSize; i += nthreads) {
        pixelIdxY = i % imageNY;
        if(pixelIdxY < imageNY -1) {
            pixelIdx = i;
            real2_type cprod = complexMulConj( image[pixelIdx], image[pixelIdx+1]);   
            phaseDiffY += cprod;
        } 
    }       
    complexSumReduceBlock<nthreads>(phaseDiffY, shmem);
    //phaseDiffY *= normCoef;
    real_type phaseY=atan2(phaseDiffY.y, phaseDiffY.x);

    real2_type phaseDiffX  = make_real2(0.0, 0.0);
    for (int i = tid; i < imageSize; i += nthreads)  {
        pixelIdxX = i / imageNY; 
        if(pixelIdxX < imageNX -1) {
            pixelIdx = i;
            real2_type cprod = complexMulConj(image[i], image[i+imageNY]);
            phaseDiffX += cprod;
        }
    }   
    
    complexSumReduceBlock<nthreads>(phaseDiffX, shmem);
   
    //phaseDiffX *= normCoef;
    real_type phaseX = atan2(phaseDiffX.y, phaseDiffX.x);  //+FLT_EPSILON
     
    for (int i = tid; i < imageSize; i += nthreads)
    { 
        pixelIdxX = i%imageNY;
        pixelIdxY = i/imageNY;
        real_type phase = pixelIdxX*phaseX + pixelIdxY*phaseY;
        real2_type phase_factor = make_real2(cosf(phase), sinf(phase));
        image[i] *= phase_factor;
    }     
}

/**
 * Deramp a complex signal with Method 1
 * @brief Each signal is decomposed into real and imaginary parts,
 *   and the average phase shift is obtained as atan(\sum imag / \sum real).
 * @param[inout] images input/output complex signals
 * @param[in] stream cuda stream
 */
void cuDerampMethod1(cuArrays<real2_type> *images, hipStream_t stream)
{
    
    const dim3 grid(images->count);
    const int imageSize = images->width*images->height;
    const real_type invSize = 1.0f/imageSize;

    if(imageSize <=64) {
        cuDerampMethod1_kernel<64> <<<grid, 64, 0, stream>>>
        (images->devData, images->height, images->width, 
        imageSize, images->count, invSize); }
     else if(imageSize <=128) {
        cuDerampMethod1_kernel<128> <<<grid, 128, 0, stream>>>
        (images->devData, images->height, images->width, 
        imageSize, images->count, invSize); }   
     else if(imageSize <=256) {
        cuDerampMethod1_kernel<256> <<<grid, 256, 0, stream>>>
        (images->devData, images->height, images->width, 
        imageSize, images->count, invSize); }  
    else  {
        cuDerampMethod1_kernel<512> <<<grid, 512, 0, stream>>>
        (images->devData, images->height, images->width, 
        imageSize, images->count, invSize); }
    getLastCudaError("cuDerampMethod1 kernel error\n");

}
        
void cuDeramp(int method, cuArrays<real2_type> *images, hipStream_t stream)
{
    switch(method) {
    case 1:
        cuDerampMethod1(images, stream);
        break;
    default:
        break;
    }
}

// end of file